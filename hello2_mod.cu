#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
               blockIdx.y * gridDim.x +
               blockIdx.x ) * blockDim.x +
               threadIdx.x;

  printf ("Hello world from %i\n", myID);
}

int main ()
{
  dim3 g (4, 3, 2);
  hello <<< g, 10 >>> ();
  hipDeviceSynchronize();
  return 0;
}
