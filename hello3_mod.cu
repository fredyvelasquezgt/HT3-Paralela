#include "hip/hip_runtime.h"
%%cu
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  // Fórmula genérica para calcular el ID global
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
               blockIdx.y * gridDim.x +
               blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
               threadIdx.z *  blockDim.x * blockDim.y +
               threadIdx.y * blockDim.x +
               threadIdx.x;

  printf ("Hello world from %i. [Fredy 201011]\n", myID); // Modificación aquí para agregar nombre y carné
}

int main ()
{
  dim3 g (4, 2);      // Modificación aquí
  dim3 b (32, 16);    // Modificación aquí

  hello <<< g, b >>> ();
  hipDeviceSynchronize();  // Usando hipDeviceSynchronize ya que hipDeviceSynchronize está obsoleto
  return 0;
}
