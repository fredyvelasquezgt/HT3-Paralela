#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   int globalID = blockIdx.x * blockDim.x + threadIdx.x; // Calcula el ID global del hilo
   if(globalID == 2047) // Sólo el último hilo imprime el mensaje especial
     printf("Hello world from thread %d of block %d. Soy Fredy y mi carnet es 201011\n", threadIdx.x, blockIdx.x);
   else
     printf("Hello world from thread %d of block %d\n", threadIdx.x, blockIdx.x);
}

int main()
{
  hello<<<2,1024>>>(); // 2 bloques, 1024 hilos por bloque
  hipDeviceSynchronize();
  return 0;
}

